#include "hip/hip_runtime.h"
/**
 * @brief  CUDA-accelerated Schulze voting alrogithm implementation.
 * @author Ash Vardanian
 * @date   July 12, 2024
 */
#include <cstdint>

#include <hip/hip_runtime.h>

#include <cub/block/block_reduce.cuh>
#include <cuda/atomic>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>

#include <pybind11/numpy.h>
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>

namespace py = pybind11;

using votes_count_t = uint32_t;
using candidate_idx_t = uint32_t;

/**
 * @brief Processes a tile of the preferences matrix for the block-parallel Schulze voting algorithm in CUDA.
 *
 * @tparam tile_size The size of the tile to be processed.
 * @tparam synchronize Whether to synchronize threads within the tile processing.
 * @tparam may_be_diagonal Whether the tile may contain diagonal elements.
 * @param c The output tile.
 * @param a The first input tile.
 * @param b The second input tile.
 * @param bi Row index within the tile.
 * @param bj Column index within the tile.
 * @param c_row Row index of the output tile in the global matrix.
 * @param c_col Column index of the output tile in the global matrix.
 * @param a_row Row index of the first input tile in the global matrix.
 * @param a_col Column index of the first input tile in the global matrix.
 * @param b_row Row index of the second input tile in the global matrix.
 * @param b_col Column index of the second input tile in the global matrix.
 */
template <uint32_t tile_size, bool synchronize = true, bool may_be_diagonal = true>
__forceinline__ __device__ void _process_tile_cuda(       //
    votes_count_t* c, votes_count_t* a, votes_count_t* b, //
    candidate_idx_t bi, candidate_idx_t bj,               //
    candidate_idx_t c_row, candidate_idx_t c_col,         //
    candidate_idx_t a_row, candidate_idx_t a_col,         //
    candidate_idx_t b_row, candidate_idx_t b_col) {

#pragma unroll
    for (candidate_idx_t k = 0; k < tile_size; k++) {
        votes_count_t smallest = umin(a[bi * tile_size + k], b[k * tile_size + bj]);
        if constexpr (may_be_diagonal) {
            uint32_t is_not_diagonal_c = (c_row + bi) != (c_col + bj);
            uint32_t is_not_diagonal_a = (a_row + bi) != (a_col + k);
            uint32_t is_not_diagonal_b = (b_row + k) != (b_col + bj);
            uint32_t is_bigger = smallest > c[bi * tile_size + bj];
            if (is_not_diagonal_c + is_not_diagonal_a + is_not_diagonal_b + is_bigger == 4)
                c[bi * tile_size + bj] = smallest;
        } else
            c[bi * tile_size + bj] = umax(c[bi * tile_size + bj], smallest);
        if constexpr (synchronize)
            __syncthreads();
    }
}

/**
 * @brief Performs the diagonal step of the block-parallel Schulze voting algorithm in CUDA.
 *
 * @tparam tile_size The size of the tile to be processed.
 * @param n The number of candidates.
 * @param k The index of the current tile being processed.
 * @param graph The graph of strongest paths.
 */
template <uint32_t tile_size>
__global__ void _step_diagonal(candidate_idx_t n, candidate_idx_t k, votes_count_t* graph) {
    candidate_idx_t const bi = threadIdx.y;
    candidate_idx_t const bj = threadIdx.x;

    __shared__ votes_count_t c[tile_size * tile_size];
    c[bi * tile_size + bj] = graph[k * tile_size * n + k * tile_size + bi * n + bj];

    __syncthreads();
    _process_tile_cuda<tile_size>(    //
        c, c, c, bi, bj,              //
        tile_size * k, tile_size * k, //
        tile_size * k, tile_size * k, //
        tile_size * k, tile_size * k  //
    );

    graph[k * tile_size * n + k * tile_size + bi * n + bj] = c[bi * tile_size + bj];
}

/**
 * @brief Performs the partially independent step of the block-parallel Schulze voting algorithm in CUDA.
 *
 * @tparam tile_size The size of the tile to be processed.
 * @param n The number of candidates.
 * @param k The index of the current tile being processed.
 * @param graph The graph of strongest paths.
 */
template <uint32_t tile_size>
__global__ void _step_partially_independent(candidate_idx_t n, candidate_idx_t k, votes_count_t* graph) {
    candidate_idx_t const i = blockIdx.x;
    candidate_idx_t const bi = threadIdx.y;
    candidate_idx_t const bj = threadIdx.x;

    if (i == k)
        return;

    __shared__ votes_count_t a[tile_size * tile_size];
    __shared__ votes_count_t b[tile_size * tile_size];
    __shared__ votes_count_t c[tile_size * tile_size];

    // Walking down within a group of adjacent columns
    c[bi * tile_size + bj] = graph[i * tile_size * n + k * tile_size + bi * n + bj];
    b[bi * tile_size + bj] = graph[k * tile_size * n + k * tile_size + bi * n + bj];

    __syncthreads();
    _process_tile_cuda<tile_size>(    //
        c, c, b, bi, bj,              //
        i * tile_size, k * tile_size, //
        i * tile_size, k * tile_size, //
        k * tile_size, k * tile_size);

    // Walking right within a group of adjacent rows
    graph[i * tile_size * n + k * tile_size + bi * n + bj] = c[bi * tile_size + bj];
    c[bi * tile_size + bj] = graph[k * tile_size * n + i * tile_size + bi * n + bj];
    a[bi * tile_size + bj] = graph[k * tile_size * n + k * tile_size + bi * n + bj];

    __syncthreads();
    _process_tile_cuda<tile_size>(    //
        c, a, c, bi, bj,              //
        k * tile_size, i * tile_size, //
        k * tile_size, k * tile_size, //
        k * tile_size, i * tile_size  //
    );

    graph[k * tile_size * n + i * tile_size + bi * n + bj] = c[bi * tile_size + bj];
}

/**
 * @brief Performs then independent step of the block-parallel Schulze voting algorithm in CUDA.
 *
 * @tparam tile_size The size of the tile to be processed.
 * @param n The number of candidates.
 * @param k The index of the current tile being processed.
 * @param graph The graph of strongest paths.
 */
template <uint32_t tile_size>
__global__ void _step_independent(candidate_idx_t n, candidate_idx_t k, votes_count_t* graph) {
    candidate_idx_t const j = blockIdx.x;
    candidate_idx_t const i = blockIdx.y;
    candidate_idx_t const bi = threadIdx.y;
    candidate_idx_t const bj = threadIdx.x;

    if (i == k && j == k)
        return;

    __shared__ votes_count_t a[tile_size * tile_size];
    __shared__ votes_count_t b[tile_size * tile_size];
    __shared__ votes_count_t c[tile_size * tile_size];

    c[bi * tile_size + bj] = graph[i * tile_size * n + j * tile_size + bi * n + bj];
    a[bi * tile_size + bj] = graph[i * tile_size * n + k * tile_size + bi * n + bj];
    b[bi * tile_size + bj] = graph[k * tile_size * n + j * tile_size + bi * n + bj];

    __syncthreads();
    if (i == j)
        // We don't need to "synchronize", because A, C, and B tile arguments
        // are different in the independent state and will address different shared buffers.
        _process_tile_cuda<tile_size, false, true>( //
            c, a, b, bi, bj,                        //
            i * tile_size, j * tile_size,           //
            i * tile_size, k * tile_size,           //
            k * tile_size, j * tile_size            //
        );
    else
        // We don't need to "synchronize", because A, C, and B tile arguments
        // are different in the independent state and will address different shared buffers.
        // We also mark as "non diagonal", because the `i != j`, and in that case
        // we can avoid some branches.
        _process_tile_cuda<tile_size, false, false>( //
            c, a, b, bi, bj,                         //
            i * tile_size, j * tile_size,            //
            i * tile_size, k * tile_size,            //
            k * tile_size, j * tile_size             //
        );

    graph[i * tile_size * n + j * tile_size + bi * n + bj] = c[bi * tile_size + bj];
}

/**
 * @brief Computes the strongest paths for the block-parallel Schulze voting algorithm in CUDA.
 *
 * @tparam tile_size The size of the tile to be processed.
 * @param preferences The preferences matrix.
 * @param num_candidates The number of candidates.
 * @param row_stride The stride between rows in the preferences matrix.
 * @param strongest_paths The output matrix of strongest paths.
 */
template <uint32_t tile_size>      //
void compute_strongest_paths_cuda( //
    votes_count_t* preferences, candidate_idx_t num_candidates, candidate_idx_t row_stride,
    votes_count_t* strongest_paths) {

#pragma omp parallel for collapse(2)
    for (candidate_idx_t i = 0; i < num_candidates; i++)
        for (candidate_idx_t j = 0; j < num_candidates; j++)
            if (i != j)
                strongest_paths[i * num_candidates + j] =
                    preferences[i * row_stride + j] > preferences[j * row_stride + i] //
                        ? preferences[i * row_stride + j]
                        : 0;

    candidate_idx_t tiles_count = (num_candidates + tile_size - 1) / tile_size;
    dim3 tile_shape(tile_size, tile_size, 1);
    dim3 independent_grid(tiles_count, tiles_count, 1);
    for (candidate_idx_t k = 0; k < tiles_count; k++) {
        _step_diagonal<tile_size><<<1, tile_shape>>>(num_candidates, k, strongest_paths);
        _step_partially_independent<tile_size><<<tiles_count, tile_shape>>>(num_candidates, k, strongest_paths);
        _step_independent<tile_size><<<independent_grid, tile_shape>>>(num_candidates, k, strongest_paths);
    }
}

/**
 * @brief Computes the strongest paths for the block-parallel Schulze voting algorithm.
 *
 * @param preferences The preferences matrix.
 * @return A NumPy array containing the strongest paths matrix.
 */
static py::array_t<votes_count_t> compute_strongest_paths(py::array_t<votes_count_t, py::array::c_style> preferences) {
    auto buf = preferences.request();
    if (buf.ndim != 2)
        throw std::runtime_error("Number of dimensions must be two");
    if (buf.shape[0] != buf.shape[1])
        throw std::runtime_error("Preferences matrix must be square");
    auto preferences_ptr = reinterpret_cast<votes_count_t*>(buf.ptr);
    auto num_candidates = static_cast<candidate_idx_t>(buf.shape[0]);
    auto row_stride = static_cast<candidate_idx_t>(buf.strides[0] / sizeof(votes_count_t));

    votes_count_t* strongest_paths_ptr = nullptr;
    hipError_t error;
    error = hipMallocManaged(&strongest_paths_ptr, num_candidates * num_candidates * sizeof(votes_count_t));
    if (error != hipSuccess)
        throw std::runtime_error("Failed to allocate memory on device");

    hipMemset(strongest_paths_ptr, 0, num_candidates * num_candidates * sizeof(votes_count_t));
    compute_strongest_paths_cuda<16>(preferences_ptr, num_candidates, row_stride, strongest_paths_ptr);

    // Synchronize to ensure all CUDA operations are complete
    error = hipDeviceSynchronize();
    if (error != hipSuccess) {
        hipFree(strongest_paths_ptr);
        throw std::runtime_error("CUDA operations did not complete successfully");
    }

    // Allocate NumPy array for the result
    auto result = py::array_t<votes_count_t>({num_candidates, num_candidates});
    auto result_buf = result.request();
    auto result_ptr = reinterpret_cast<votes_count_t*>(result_buf.ptr);

    // Copy data from the GPU to the NumPy array
    error = hipMemcpy(result_ptr, strongest_paths_ptr, num_candidates * num_candidates * sizeof(votes_count_t),
                       hipMemcpyDeviceToHost);
    if (error != hipSuccess) {
        hipFree(strongest_paths_ptr);
        throw std::runtime_error("Failed to copy data from device to host");
    }

    // Synchronize to ensure all CUDA transfers are complete
    error = hipDeviceSynchronize();
    if (error != hipSuccess) {
        hipFree(strongest_paths_ptr);
        throw std::runtime_error("CUDA transfers did not complete successfully");
    }

    // Free the GPU memory
    error = hipFree(strongest_paths_ptr);
    if (error != hipSuccess)
        throw std::runtime_error("Failed to free memory on device");

    return result;
}

PYBIND11_MODULE(scaling_democracy, m) {

    // Let's show how to wrap `void` functions for basic logging
    m.def("log_devices", []() {
        int deviceCount;
        hipGetDeviceCount(&deviceCount);
        for (int i = 0; i < deviceCount; i++) {
            hipDeviceProp_t deviceProps;
            hipGetDeviceProperties(&deviceProps, i);
            printf("Device %d: %s\n", i, deviceProps.name);
            printf("\tSMs: %d\n", deviceProps.multiProcessorCount);
            printf("\tGlobal mem: %.2fGB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024 * 1024));
            printf("\tCUDA Cap: %d.%d\n", deviceProps.major, deviceProps.minor);
        }
    });

    // This is how we could have used `thrust::` for higher-level operations
    m.def("reduce", [](py::array_t<float> const& data) -> float {
        py::buffer_info buf = data.request();
        if (buf.ndim != 1 || buf.strides[0] != sizeof(float))
            throw std::runtime_error("Input should be a contiguous 1D float array");
        float* ptr = static_cast<float*>(buf.ptr);
        thrust::device_vector<float> d_data(ptr, ptr + buf.size);
        return thrust::reduce(thrust::device, d_data.begin(), d_data.end(), 0.0f);
    });

    m.def("compute_strongest_paths", &compute_strongest_paths);
}
