#include "hip/hip_runtime.h"
/**
 * @brief  CUDA-accelerated Schulze voting alrogithm implementation.
 * @author Ash Vardanian
 * @date   July 12, 2024
 */
#include <cstdint>

#include <hip/hip_runtime.h>

#include <hip/hip_runtime.h> // `CUtensorMap`
#include <cuda/barrier>
#include <cudaTypedefs.h> // `PFN_cuTensorMapEncodeTiled`
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>

#include <pybind11/numpy.h> // `array_t`
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 300
#define SCALING_DEMOCRACY_KEPLER 1
#endif
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 900
#define SCALING_DEMOCRACY_HOPPER 1
#endif

namespace py = pybind11;
namespace cde = cuda::device::experimental;
using barrier_t = cuda::barrier<cuda::thread_scope_block>;

using votes_count_t = uint32_t;
using candidate_idx_t = uint32_t;

#if defined(SCALING_DEMOCRACY_KEPLER)

/**
 * @brief   Processes a tile of the preferences matrix for the block-parallel Schulze voting algorithm
 *          in CUDA on Nvidia @b Kepler GPUs and newer (sm_30).
 *
 * @tparam tile_size The size of the tile to be processed.
 * @tparam synchronize Whether to synchronize threads within the tile processing.
 * @tparam may_be_diagonal Whether the tile may contain diagonal elements.
 * @param c The output tile.
 * @param a The first input tile.
 * @param b The second input tile.
 * @param bi Row index within the tile.
 * @param bj Column index within the tile.
 * @param c_row Row index of the output tile in the global matrix.
 * @param c_col Column index of the output tile in the global matrix.
 * @param a_row Row index of the first input tile in the global matrix.
 * @param a_col Column index of the first input tile in the global matrix.
 * @param b_row Row index of the second input tile in the global matrix.
 * @param b_col Column index of the second input tile in the global matrix.
 */
template <uint32_t tile_size, bool synchronize = true, bool may_be_diagonal = true>
__forceinline__ __device__ void _process_tile_cuda(                   //
    votes_count_t* c, votes_count_t const* a, votes_count_t const* b, //
    candidate_idx_t bi, candidate_idx_t bj,                           //
    candidate_idx_t c_row, candidate_idx_t c_col,                     //
    candidate_idx_t a_row, candidate_idx_t a_col,                     //
    candidate_idx_t b_row, candidate_idx_t b_col) {

#pragma unroll(tile_size)
    for (candidate_idx_t k = 0; k < tile_size; k++) {
        votes_count_t smallest = umin(a[bi * tile_size + k], b[k * tile_size + bj]);
        if constexpr (may_be_diagonal) {
            uint32_t is_not_diagonal_c = (c_row + bi) != (c_col + bj);
            uint32_t is_not_diagonal_a = (a_row + bi) != (a_col + k);
            uint32_t is_not_diagonal_b = (b_row + k) != (b_col + bj);
            uint32_t is_bigger = smallest > c[bi * tile_size + bj];
            uint32_t will_replace = is_not_diagonal_c & is_not_diagonal_a & is_not_diagonal_b & is_bigger;
            // On Kepler an newer we can use `__funnelshift_lc` to avoid branches
            c[bi * tile_size + bj] = __funnelshift_lc(c[bi * tile_size + bj], smallest, will_replace - 1);
        } else
            c[bi * tile_size + bj] = umax(c[bi * tile_size + bj], smallest);
        if constexpr (synchronize)
            __syncthreads();
    }
}

#else

/**
 * @brief   Processes a tile of the preferences matrix for the block-parallel Schulze voting algorithm
 *          in CUDA or @b HIP.
 *
 * @tparam tile_size The size of the tile to be processed.
 * @tparam synchronize Whether to synchronize threads within the tile processing.
 * @tparam may_be_diagonal Whether the tile may contain diagonal elements.
 * @param c The output tile.
 * @param a The first input tile.
 * @param b The second input tile.
 * @param bi Row index within the tile.
 * @param bj Column index within the tile.
 * @param c_row Row index of the output tile in the global matrix.
 * @param c_col Column index of the output tile in the global matrix.
 * @param a_row Row index of the first input tile in the global matrix.
 * @param a_col Column index of the first input tile in the global matrix.
 * @param b_row Row index of the second input tile in the global matrix.
 * @param b_col Column index of the second input tile in the global matrix.
 */
template <uint32_t tile_size, bool synchronize = true, bool may_be_diagonal = true>
__forceinline__ __device__ void _process_tile_cuda(                   //
    votes_count_t* c, votes_count_t const* a, votes_count_t const* b, //
    candidate_idx_t bi, candidate_idx_t bj,                           //
    candidate_idx_t c_row, candidate_idx_t c_col,                     //
    candidate_idx_t a_row, candidate_idx_t a_col,                     //
    candidate_idx_t b_row, candidate_idx_t b_col) {

#pragma unroll(tile_size)
    for (candidate_idx_t k = 0; k < tile_size; k++) {
        votes_count_t smallest = min(a[bi * tile_size + k], b[k * tile_size + bj]);
        if constexpr (may_be_diagonal) {
            uint32_t is_not_diagonal_c = (c_row + bi) != (c_col + bj);
            uint32_t is_not_diagonal_a = (a_row + bi) != (a_col + k);
            uint32_t is_not_diagonal_b = (b_row + k) != (b_col + bj);
            uint32_t is_bigger = smallest > c[bi * tile_size + bj];
            uint32_t will_replace = is_not_diagonal_c & is_not_diagonal_a & is_not_diagonal_b & is_bigger;
            if (will_replace)
                c[bi * tile_size + bj] = smallest;
        } else
            c[bi * tile_size + bj] = max(c[bi * tile_size + bj], min(a[bi * tile_size + k], b[k * tile_size + bj]));
        if constexpr (synchronize)
            __syncthreads();
    }
}

#endif

/**
 * @brief Performs the diagonal step of the block-parallel Schulze voting algorithm in CUDA or @b HIP.
 *
 * @tparam tile_size The size of the tile to be processed.
 * @param n The number of candidates.
 * @param k The index of the current tile being processed.
 * @param graph The graph of strongest paths.
 */
template <uint32_t tile_size>
__global__ void _step_diagonal(candidate_idx_t n, candidate_idx_t k, votes_count_t* graph) {
    candidate_idx_t const bi = threadIdx.y;
    candidate_idx_t const bj = threadIdx.x;

    __shared__ alignas(16) votes_count_t c[tile_size * tile_size];
    c[bi * tile_size + bj] = graph[k * tile_size * n + k * tile_size + bi * n + bj];

    __syncthreads();
    _process_tile_cuda<tile_size>(    //
        c, c, c, bi, bj,              //
        tile_size * k, tile_size * k, //
        tile_size * k, tile_size * k, //
        tile_size * k, tile_size * k  //
    );

    graph[k * tile_size * n + k * tile_size + bi * n + bj] = c[bi * tile_size + bj];
}

/**
 * @brief Performs the partially independent step of the block-parallel Schulze voting algorithm in CUDA or @b HIP.
 *
 * @tparam tile_size The size of the tile to be processed.
 * @param n The number of candidates.
 * @param k The index of the current tile being processed.
 * @param graph The graph of strongest paths.
 */
template <uint32_t tile_size>
__global__ void _step_partially_independent(candidate_idx_t n, candidate_idx_t k, votes_count_t* graph) {
    candidate_idx_t const i = blockIdx.x;
    candidate_idx_t const bi = threadIdx.y;
    candidate_idx_t const bj = threadIdx.x;

    if (i == k)
        return;

    __shared__ alignas(16) votes_count_t a[tile_size * tile_size];
    __shared__ alignas(16) votes_count_t b[tile_size * tile_size];
    __shared__ alignas(16) votes_count_t c[tile_size * tile_size];

    // Walking down within a group of adjacent columns
    c[bi * tile_size + bj] = graph[i * tile_size * n + k * tile_size + bi * n + bj];
    b[bi * tile_size + bj] = graph[k * tile_size * n + k * tile_size + bi * n + bj];

    __syncthreads();
    _process_tile_cuda<tile_size>(    //
        c, c, b, bi, bj,              //
        i * tile_size, k * tile_size, //
        i * tile_size, k * tile_size, //
        k * tile_size, k * tile_size);

    // Walking right within a group of adjacent rows
    __syncthreads();
    graph[i * tile_size * n + k * tile_size + bi * n + bj] = c[bi * tile_size + bj];
    c[bi * tile_size + bj] = graph[k * tile_size * n + i * tile_size + bi * n + bj];
    a[bi * tile_size + bj] = graph[k * tile_size * n + k * tile_size + bi * n + bj];

    __syncthreads();
    _process_tile_cuda<tile_size>(    //
        c, a, c, bi, bj,              //
        k * tile_size, i * tile_size, //
        k * tile_size, k * tile_size, //
        k * tile_size, i * tile_size  //
    );

    graph[k * tile_size * n + i * tile_size + bi * n + bj] = c[bi * tile_size + bj];
}

/**
 * @brief Performs then independent step of the block-parallel Schulze voting algorithm in CUDA or @b HIP.
 *
 * @tparam tile_size The size of the tile to be processed.
 * @param n The number of candidates.
 * @param k The index of the current tile being processed.
 * @param graph The graph of strongest paths.
 */
template <uint32_t tile_size>
__global__ void _step_independent(candidate_idx_t n, candidate_idx_t k, votes_count_t* graph) {
    candidate_idx_t const j = blockIdx.x;
    candidate_idx_t const i = blockIdx.y;
    candidate_idx_t const bi = threadIdx.y;
    candidate_idx_t const bj = threadIdx.x;

    if (i == k && j == k)
        return;

    __shared__ alignas(16) votes_count_t a[tile_size * tile_size];
    __shared__ alignas(16) votes_count_t b[tile_size * tile_size];
    __shared__ alignas(16) votes_count_t c[tile_size * tile_size];

    c[bi * tile_size + bj] = graph[i * tile_size * n + j * tile_size + bi * n + bj];
    a[bi * tile_size + bj] = graph[i * tile_size * n + k * tile_size + bi * n + bj];
    b[bi * tile_size + bj] = graph[k * tile_size * n + j * tile_size + bi * n + bj];

    __syncthreads();
    if (i == j)
        // We don't need to "synchronize", because A, C, and B tile arguments
        // are different in the independent state and will address different shared buffers.
        _process_tile_cuda<tile_size, false, true>( //
            c, a, b, bi, bj,                        //
            i * tile_size, j * tile_size,           //
            i * tile_size, k * tile_size,           //
            k * tile_size, j * tile_size            //
        );
    else
        // We don't need to "synchronize", because A, C, and B tile arguments
        // are different in the independent state and will address different shared buffers.
        // We also mark as "non diagonal", because the `i != j`, and in that case
        // we can avoid some branches.
        _process_tile_cuda<tile_size, false, false>( //
            c, a, b, bi, bj,                         //
            i * tile_size, j * tile_size,            //
            i * tile_size, k * tile_size,            //
            k * tile_size, j * tile_size             //
        );

    graph[i * tile_size * n + j * tile_size + bi * n + bj] = c[bi * tile_size + bj];
}

/**
 * @brief Performs then independent step of the block-parallel Schulze voting algorithm in CUDA or @b HIP.
 *
 * @tparam tile_size The size of the tile to be processed.
 * @param n The number of candidates.
 * @param k The index of the current tile being processed.
 * @param graph The graph of strongest paths represented as a `CUtensorMap`.
 */
template <uint32_t tile_size>
__global__ void _step_independent_hopper(candidate_idx_t n, candidate_idx_t k,
                                         __grid_constant__ CUtensorMap const graph) {
    candidate_idx_t const j = blockIdx.x;
    candidate_idx_t const i = blockIdx.y;
    candidate_idx_t const bi = threadIdx.y;
    candidate_idx_t const bj = threadIdx.x;

#if defined(SCALING_DEMOCRACY_HOPPER)

    if (i == k && j == k)
        return;

    __shared__ alignas(128) votes_count_t a[tile_size][tile_size];
    __shared__ alignas(128) votes_count_t b[tile_size][tile_size];
    __shared__ alignas(128) votes_count_t c[tile_size][tile_size];

#pragma nv_diag_suppress static_var_with_dynamic_init
    // Initialize shared memory barrier with the number of threads participating in the barrier.
    __shared__ barrier_t bar;
    if (threadIdx.x == 0) {
        // We have one thread per tile cell.
        init(&bar, tile_size * tile_size);
        // Make initialized barrier visible in async proxy.
        cde::fence_proxy_async_shared_cta();
    }
    // Syncthreads so initialized barrier is visible to all threads.
    __syncthreads();

    // Only the first thread in the tile invokes the bulk transfers.
    barrier_t::arrival_token token;
    if (threadIdx.x == 0) {
        // Initiate three bulk tensor copies for different part of the graph.
        cde::cp_async_bulk_tensor_2d_global_to_shared(&c, &graph, i * tile_size, j * tile_size, bar);
        cde::cp_async_bulk_tensor_2d_global_to_shared(&a, &graph, i * tile_size, k * tile_size, bar);
        cde::cp_async_bulk_tensor_2d_global_to_shared(&b, &graph, k * tile_size, j * tile_size, bar);
        // Arrive on the barrier and tell how many bytes are expected to come in.
        token = cuda::device::barrier_arrive_tx(bar, 1, sizeof(c) + sizeof(a) + sizeof(b));
    } else {
        // Other threads just arrive.
        token = bar.arrive(1);
    }

    // Wait for the data to have arrived.
    // After this point we expect shared memory to contain the following data:
    //
    //  c[bi * tile_size + bj] = graph[i * tile_size * n + j * tile_size + bi * n + bj];
    //  a[bi * tile_size + bj] = graph[i * tile_size * n + k * tile_size + bi * n + bj];
    //  b[bi * tile_size + bj] = graph[k * tile_size * n + j * tile_size + bi * n + bj];
    bar.wait(std::move(token));

    if (i == j)
        // We don't need to "synchronize", because A, C, and B tile arguments
        // are different in the independent state and will address different shared buffers.
        _process_tile_cuda<tile_size, false, true>( //
            &c[0][0], &a[0][0], &b[0][0], bi, bj,   //
            i * tile_size, j * tile_size,           //
            i * tile_size, k * tile_size,           //
            k * tile_size, j * tile_size            //
        );
    else
        // We don't need to "synchronize", because A, C, and B tile arguments
        // are different in the independent state and will address different shared buffers.
        // We also mark as "non diagonal", because the `i != j`, and in that case
        // we can avoid some branches.
        _process_tile_cuda<tile_size, false, false>( //
            &c[0][0], &a[0][0], &b[0][0], bi, bj,    //
            i * tile_size, j * tile_size,            //
            i * tile_size, k * tile_size,            //
            k * tile_size, j * tile_size             //
        );

    // Wait for shared memory writes to be visible to TMA engine.
    cde::fence_proxy_async_shared_cta();
    __syncthreads();
    // After syncthreads, writes by all threads are visible to TMA engine.

    // Initiate TMA transfer to copy shared memory to global memory
    if (threadIdx.x == 0) {
        cde::cp_async_bulk_tensor_2d_shared_to_global(&graph, i * tile_size, j * tile_size, &c);
        // Wait for TMA transfer to have finished reading shared memory.
        // Create a "bulk async-group" out of the previous bulk copy operation.
        cde::cp_async_bulk_commit_group();
        // Wait for the group to have completed reading from shared memory.
        cde::cp_async_bulk_wait_group_read<0>();

        // Destroy barrier. This invalidates the memory region of the barrier. If
        // further computations were to take place in the kernel, this allows the
        // memory location of the shared memory barrier to be reused.
        (&bar)->~barrier();
    }
#else
    // This is a trap :)
    if (i == 0 && j == 0 && bi == 0 && bj == 0)
        printf("This kernel is only supported on Hopper and newer GPUs\n");
#endif
}

PFN_cuTensorMapEncodeTiled_v12000 get_cuTensorMapEncodeTiled() {
    // Get pointer to hipGetProcAddress
    hipDriverEntryPointQueryResult driver_status;
    void* cuGetProcAddress_ptr = nullptr;
    hipError_t error =
        hipGetDriverEntryPoint("hipGetProcAddress", &cuGetProcAddress_ptr, hipEnableDefault, &driver_status);
    if (error != hipSuccess)
        throw std::runtime_error("Failed to get hipGetProcAddress");
    if (driver_status != hipDriverEntryPointSuccess)
        throw std::runtime_error("Failed to get hipGetProcAddress entry point");
    PFN_cuGetProcAddress_v12000 hipGetProcAddress = reinterpret_cast<PFN_cuGetProcAddress_v12000>(cuGetProcAddress_ptr);

    // Use hipGetProcAddress to get a pointer to the CTK 12.0 version of cuTensorMapEncodeTiled
    hipDriverProcAddressQueryResult symbol_status;
    void* cuTensorMapEncodeTiled_ptr = nullptr;
    hipError_t res = hipGetProcAddress("cuTensorMapEncodeTiled", &cuTensorMapEncodeTiled_ptr, 12000,
                                    hipEnableDefault, &symbol_status);
    if (res != hipSuccess || symbol_status != HIP_GET_PROC_ADDRESS_SUCCESS)
        throw std::runtime_error("Failed to get cuTensorMapEncodeTiled");
    return reinterpret_cast<PFN_cuTensorMapEncodeTiled_v12000>(cuTensorMapEncodeTiled_ptr);
}

/**
 * @brief Computes the strongest paths for the block-parallel Schulze voting algorithm in CUDA or @b HIP.
 *
 * @tparam tile_size The size of the tile to be processed.
 * @param preferences The preferences matrix.
 * @param num_candidates The number of candidates.
 * @param row_stride The stride between rows in the preferences matrix.
 * @param strongest_paths The output matrix of strongest paths.
 */
template <uint32_t tile_size>      //
void compute_strongest_paths_cuda( //
    votes_count_t* preferences, candidate_idx_t num_candidates, candidate_idx_t row_stride,
    votes_count_t* strongest_paths) {

#pragma omp parallel for collapse(2)
    for (candidate_idx_t i = 0; i < num_candidates; i++)
        for (candidate_idx_t j = 0; j < num_candidates; j++)
            if (i != j)
                strongest_paths[i * num_candidates + j] =
                    preferences[i * row_stride + j] > preferences[j * row_stride + i] //
                        ? preferences[i * row_stride + j]
                        : 0;

    // Check if we can use newer CUDA features.
    hipError_t error;
    int current_device;
    hipDeviceProp_t device_props;
    error = hipGetDevice(&current_device);
    if (error != hipSuccess)
        throw std::runtime_error("Failed to get current device");
    error = hipGetDeviceProperties(&device_props, current_device);
    if (error != hipSuccess)
        throw std::runtime_error("Failed to get device properties");
    bool supports_tma = device_props.major >= 9;
    printf("Device %d supports TMA: %s\n", current_device, supports_tma ? "yes" : "no");

    CUtensorMap strongest_paths_tensor_map{};
    // rank is the number of dimensions of the array.
    constexpr uint32_t rank = 2;
    uint64_t size[rank] = {num_candidates, num_candidates};
    // The stride is the number of bytes to traverse from the first element of one row to the next.
    // It must be a multiple of 16.
    uint64_t stride[rank - 1] = {num_candidates * sizeof(votes_count_t)};
    // The box_size is the size of the shared memory buffer that is used as the
    // destination of a TMA transfer.
    uint32_t box_size[rank] = {tile_size, tile_size};
    // The distance between elements in units of sizeof(element). A stride of 2
    // can be used to load only the real component of a complex-valued tensor, for instance.
    uint32_t elem_stride[rank] = {1, 1};

    // Create the tensor descriptor.
    // https://docs.nvidia.com/cuda/cuda-driver-api/group__CUDA__TENSOR__MEMORY.html#group__CUDA__TENSOR__MEMORY_1ga7c7d2aaac9e49294304e755e6f341d7
    PFN_cuTensorMapEncodeTiled_v12000 cuTensorMapEncodeTiled = get_cuTensorMapEncodeTiled();
    hipError_t res = cuTensorMapEncodeTiled( //
        &strongest_paths_tensor_map,       // CUtensorMap *tensorMap,
        CUtensorMapDataType::CU_TENSOR_MAP_DATA_TYPE_UINT32,
        rank,            // cuuint32_t tensorRank,
        strongest_paths, // void *globalAddress,
        size,            // const cuuint64_t *globalDim,
        stride,          // const cuuint64_t *globalStrides,
        box_size,        // const cuuint32_t *boxDim,
        elem_stride,     // const cuuint32_t *elementStrides,
        // Interleave patterns can be used to accelerate loading of values that
        // are less than 4 bytes long.
        CUtensorMapInterleave::CU_TENSOR_MAP_INTERLEAVE_NONE,
        // Swizzling can be used to avoid shared memory bank conflicts.
        CUtensorMapSwizzle::CU_TENSOR_MAP_SWIZZLE_NONE,
        // L2 Promotion can be used to widen the effect of a cache-policy to a wider
        // set of L2 cache lines. Can be 64b, 128b, 256b, or none.
        CUtensorMapL2promotion::CU_TENSOR_MAP_L2_PROMOTION_L2_256B,
        // Any element that is outside of bounds will be set to zero by the TMA transfer.
        CUtensorMapFloatOOBfill::CU_TENSOR_MAP_FLOAT_OOB_FILL_NONE);

    candidate_idx_t tiles_count = (num_candidates + tile_size - 1) / tile_size;
    dim3 tile_shape(tile_size, tile_size, 1);
    dim3 independent_grid(tiles_count, tiles_count, 1);
    for (candidate_idx_t k = 0; k < tiles_count; k++) {
        _step_diagonal<tile_size><<<1, tile_shape>>>(num_candidates, k, strongest_paths);
        _step_partially_independent<tile_size><<<tiles_count, tile_shape>>>(num_candidates, k, strongest_paths);
        if (supports_tma)
            _step_independent_hopper<tile_size>
                <<<independent_grid, tile_shape>>>(num_candidates, k, strongest_paths_tensor_map);
        else
            _step_independent<tile_size><<<independent_grid, tile_shape>>>(num_candidates, k, strongest_paths);

        error = hipGetLastError();
        if (error != hipSuccess)
            throw std::runtime_error(hipGetErrorString(error));
    }
}

/**
 * @brief Computes the strongest paths for the block-parallel Schulze voting algorithm.
 *
 * @param preferences The preferences matrix.
 * @return A NumPy array containing the strongest paths matrix.
 */
static py::array_t<votes_count_t> compute_strongest_paths(py::array_t<votes_count_t, py::array::c_style> preferences) {
    auto buf = preferences.request();
    if (buf.ndim != 2)
        throw std::runtime_error("Number of dimensions must be two");
    if (buf.shape[0] != buf.shape[1])
        throw std::runtime_error("Preferences matrix must be square");
    auto preferences_ptr = reinterpret_cast<votes_count_t*>(buf.ptr);
    auto num_candidates = static_cast<candidate_idx_t>(buf.shape[0]);
    auto row_stride = static_cast<candidate_idx_t>(buf.strides[0] / sizeof(votes_count_t));

    votes_count_t* strongest_paths_ptr = nullptr;
    hipError_t error;
    error = hipMallocManaged(&strongest_paths_ptr, num_candidates * num_candidates * sizeof(votes_count_t));
    if (error != hipSuccess)
        throw std::runtime_error("Failed to allocate memory on device");

    hipMemset(strongest_paths_ptr, 0, num_candidates * num_candidates * sizeof(votes_count_t));
    compute_strongest_paths_cuda<16>(preferences_ptr, num_candidates, row_stride, strongest_paths_ptr);

    // Synchronize to ensure all CUDA operations are complete
    error = hipDeviceSynchronize();
    if (error != hipSuccess) {
        hipFree(strongest_paths_ptr);
        throw std::runtime_error("CUDA operations did not complete successfully");
    }

    // Allocate NumPy array for the result
    auto result = py::array_t<votes_count_t>({num_candidates, num_candidates});
    auto result_buf = result.request();
    auto result_ptr = reinterpret_cast<votes_count_t*>(result_buf.ptr);

    // Copy data from the GPU to the NumPy array
    error = hipMemcpy(result_ptr, strongest_paths_ptr, num_candidates * num_candidates * sizeof(votes_count_t),
                       hipMemcpyDeviceToHost);
    if (error != hipSuccess) {
        hipFree(strongest_paths_ptr);
        throw std::runtime_error("Failed to copy data from device to host");
    }

    // Synchronize to ensure all CUDA transfers are complete
    error = hipDeviceSynchronize();
    if (error != hipSuccess) {
        hipFree(strongest_paths_ptr);
        throw std::runtime_error("CUDA transfers did not complete successfully");
    }

    // Free the GPU memory
    error = hipFree(strongest_paths_ptr);
    if (error != hipSuccess)
        throw std::runtime_error("Failed to free memory on device");

    return result;
}

PYBIND11_MODULE(scaling_democracy, m) {

    // Let's show how to wrap `void` functions for basic logging
    m.def("log_devices", []() {
        int device_count;
        hipDeviceProp_t device_props;
        hipError_t error = hipGetDeviceCount(&device_count);
        if (error != hipSuccess)
            throw std::runtime_error("Failed to get device count");
        for (int i = 0; i < device_count; i++) {
            error = hipGetDeviceProperties(&device_props, i);
            if (error != hipSuccess)
                throw std::runtime_error("Failed to get device properties");
            printf("Device %d: %s\n", i, device_props.name);
            printf("\tSMs: %d\n", device_props.multiProcessorCount);
            printf("\tGlobal mem: %.2fGB\n", static_cast<float>(device_props.totalGlobalMem) / (1024 * 1024 * 1024));
            printf("\tCUDA Cap: %d.%d\n", device_props.major, device_props.minor);
        }
    });

    // This is how we could have used `thrust::` for higher-level operations
    m.def("reduce", [](py::array_t<float> const& data) -> float {
        py::buffer_info buf = data.request();
        if (buf.ndim != 1 || buf.strides[0] != sizeof(float))
            throw std::runtime_error("Input should be a contiguous 1D float array");
        float* ptr = static_cast<float*>(buf.ptr);
        thrust::device_vector<float> d_data(ptr, ptr + buf.size);
        return thrust::reduce(thrust::device, d_data.begin(), d_data.end(), 0.0f);
    });

    m.def("compute_strongest_paths", &compute_strongest_paths);
}
